#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>

#define max_N 100000
#define max_P 30
#define BLOCKSIZE 1024

using namespace std;

//*******************************************
// This kernel counts the number of conflicting requests in a facility ans store it in d_confreqs array
__global__ void countkernell(int* d_req_cen,int* d_req_fac,int* d_confreqs,int R){
      int id=blockIdx.x*1024+threadIdx.x*32+threadIdx.y;
      if(id<R){
        int offset=d_req_cen[id]*max_P+d_req_fac[id];
        atomicAdd(d_confreqs+offset,1);
      }
}
// This kernel sets the capacity value for all slots of each facility in d_slotspercenfac array
__global__ void setarraykernel(int* d_slotspercenfac,int* d_capacity,int N){
    int id=blockIdx.x*1024+threadIdx.x*32+threadIdx.y;
    if(id<N*max_P*24){
        int cen=id/(max_P*24);
        int fac=(id%(max_P*24))/24;
        d_slotspercenfac[id]=d_capacity[cen*max_P+fac];
    }
}
// This kernel processes requests on the  basis of request id for same facility and centre
__global__ void processRequests(int R,  int* req_cen, int* req_fac, int* req_start,
 int* req_slots, int* d_prefix, int* capacity, int* succ_reqs, int* tot_reqs,int *d_succ_cenreqs, int N)
{
    int tid=blockIdx.x*1024+threadIdx.x*32+threadIdx.y;
    // process requests if its a valid facility
    if(tid < N*max_P)
    {
    
        int startreq,endreq;   // startreq, endreq stores the start, end of number of request for that facility
        // Extract request details
        int cen_val = tid/max_P;
        int fac_val = tid%max_P;
        int uniqid = cen_val * max_P + fac_val;    // uniqid stores unique index of facility for this request
        int slot_idx = uniqid*24;      // slot_idx stores the starting index of capacity array for that request
        int slots_available;
        if(uniqid == 0)      // if uniqid is 0 startreq will be zero and the previos prefix array value otherwise
            startreq = 0;
        else
            startreq=d_prefix[uniqid-1];

        endreq=d_prefix[uniqid];
        for(int i=startreq;i<endreq;i++){       //process request for this facility
            slots_available=0;
            int slot_start_val = req_start[i];
            int req_slots_val = req_slots[i];
            // Check if requested slots are available
            for (int i = slot_start_val-1; i < slot_start_val + req_slots_val -1 ; i++)
            {
                //if slot is available count it
                if (capacity[slot_idx + i] > 0)
                {
                    slots_available++;
                }
            }
            // If all requested slots are available, mark request as successful
            if (slots_available == req_slots_val)
            {
                // Decrement capacity for the granted request
                for (int i = slot_start_val-1; i < slot_start_val + req_slots_val -1; i++)
                {
                    capacity[slot_idx + i]--;
                }
                atomicAdd(&d_succ_cenreqs[cen_val], 1);    //atomically increment succesfull request for this centre
                atomicAdd(succ_reqs,1);       //atomically increment succesfull request in all
            }
            atomicAdd(&tot_reqs[cen_val], 1);      //atomically increment this in number of requests
        }
    }
}
int main(int argc,char **argv)
{
	// variable declarations...
    int N,*centre,*facility,*capacity,*d_capacity, *fac_ids, *succ_cenreqs, *tot_reqs;
    FILE *inputfilepointer;
    
    //File Opening for read
    char *inputfilename = argv[1];
    inputfilepointer    = fopen( inputfilename , "r");

    if ( inputfilepointer == NULL )  {
        printf( "input.txt file failed to open." );
        return 0; 
    }

    fscanf( inputfilepointer, "%d", &N ); // N is number of centres
	
    // Allocate memory on cpu
    centre=(int*)malloc(N * sizeof (int));  // Computer  centre numbers
    facility=(int*)malloc(N * sizeof (int));  // Number of facilities in each computer centre
    fac_ids=(int*)malloc(max_P * N  * sizeof (int));  // Facility room numbers of each computer centre
    capacity=(int*)malloc(max_P * N * sizeof (int));  // stores capacities of each facility for every computer centre 
    memset(capacity,0,max_P*N*sizeof(int));
    hipMalloc(&d_capacity, N*max_P*sizeof(int));   ///allocating memory for capacity array in GPU
    
    int success,fail;  // total successful,failed requests
    int *succ_reqs=(int*)malloc(sizeof (int));  //variable in CPU
    int *d_succ_reqs;
    hipMalloc(&d_succ_reqs, sizeof(int));     //variable in GPU
    hipMemset(d_succ_reqs,0,sizeof(int));

    tot_reqs = (int *)malloc(N*sizeof(int));   // total requests for each centre
    succ_cenreqs = (int *)malloc(N*sizeof(int)); // total successful requests for each centre
    int* d_tot_reqs,*d_succ_cenreqs;   //device arrays
    hipMalloc(&d_tot_reqs, N*sizeof(int));
    hipMemset(d_tot_reqs,0,sizeof(int));
    hipMalloc(&d_succ_cenreqs, N*sizeof(int));
    hipMemset(d_succ_cenreqs,0,sizeof(int));

    // Input the computer centres data
    int k1=0;
    for(int i=0;i<N;i++)
    {
      fscanf( inputfilepointer, "%d", &centre[i] );
      fscanf( inputfilepointer, "%d", &facility[i] );
      
      for(int j=0;j<facility[i];j++)
      {
        fscanf( inputfilepointer, "%d", &fac_ids[k1] );
        k1++;
      }
      for(int j=0;j<facility[i];j++)
      {
        fscanf( inputfilepointer, "%d", &capacity[centre[i]*max_P+j]);  //setting capacity of each facility for each centre in capacity array
      }
    }
    hipMemcpy(d_capacity, capacity, N*max_P*sizeof(int), hipMemcpyHostToDevice);  //setting value in d_capacity array
    int *req_id, *req_cen, *req_fac, *req_start, *req_slots,*d_req_cen,*d_req_fac;   // Number of slots requested for every request
    int *d_confreqs,*confreqs, *d_prefix, *prefix, *d_slotspercenfac; //d_prefix: device arr which counts the starting index of requests to same computer centre and facility room number
    // Allocate memory on CPU 
	int R;
	fscanf( inputfilepointer, "%d", &R); // Total requests
    req_id = (int *) malloc ( (R) * sizeof (int) );  // Request ids
    req_cen = (int *) malloc ( (R) * sizeof (int) );  // Requested computer centre
    req_fac = (int *) malloc ( (R) * sizeof (int) );  // Requested facility
    req_start = (int *) malloc ( (R) * sizeof (int) );  // Start slot of every request
    req_slots = (int *) malloc ( (R) * sizeof (int) );   // Number of slots requested for every request
    confreqs = (int *) malloc ( (N * max_P) * sizeof (int) ); // counts request to same facility
    prefix = (int *) malloc ( (N * max_P) * sizeof (int) );   // stores the starting index of requests on a particular facility

    //Allocate memory on GPU
    hipMalloc(&d_slotspercenfac, N*max_P*24*sizeof(int));
    hipMalloc(&d_confreqs, N*max_P*sizeof(int));
    hipMalloc(&d_prefix, N*max_P*sizeof(int));
    hipMalloc(&d_req_cen, R*sizeof(int));
    hipMalloc(&d_req_fac, R*sizeof(int));
    
    // Input the user request data
    for(int j = 0; j < R; j++)
    {
       fscanf( inputfilepointer, "%d", &req_id[j]);
       fscanf( inputfilepointer, "%d", &req_cen[j]);
       fscanf( inputfilepointer, "%d", &req_fac[j]);
       fscanf( inputfilepointer, "%d", &req_start[j]);
       fscanf( inputfilepointer, "%d", &req_slots[j]);
       tot_reqs[req_cen[j]]+=1;  
    }
    //set and copy values into the d_array
    hipMemcpy(d_req_cen, req_cen, R*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_req_fac, req_fac, R*sizeof(int), hipMemcpyHostToDevice);
    hipMemset(d_confreqs, 0, N*max_P*sizeof(int));
    hipMemset(d_prefix, 0, N*max_P*sizeof(int));
    hipMemset(d_slotspercenfac, 0, N*max_P*24*sizeof(int));
	// Create an index array to keep track of the original indices
    int index[R];
    for (int i = 0; i < R; ++i) {
        index[i] = i;
    }
    // sort this index array according to centre id then facility id and then request id
    std::sort(index, index + R, [&](int a, int b) {
        if (req_cen[a] != req_cen[b]) {
            return req_cen[a] < req_cen[b];
        } else if (req_fac[a] != req_fac[b]) {
            return req_fac[a] < req_fac[b];
        } else {
            return req_id[a] < req_id[b];
        }
    });
    //sorted request variables
    int *req_cen_sorted,*req_fac_sorted,*req_start_sorted,*req_slots_sorted,*req_id_sorted;
    req_cen_sorted = (int *) malloc ( (R) * sizeof (int) );
    req_fac_sorted = (int *) malloc ( (R) * sizeof (int) );
    req_start_sorted = (int *) malloc ( (R) * sizeof (int) );
    req_slots_sorted = (int *) malloc ( (R) * sizeof (int) );
    req_id_sorted = (int *) malloc ( (R) * sizeof (int) );
    // Rearrange the input arrays based on the sorted indices
    for (int i = 0; i < R; ++i)
    {
        int idx = index[i];
        req_cen_sorted[i] = req_cen[idx];
        req_fac_sorted[i] = req_fac[idx];
        req_start_sorted[i] = req_start[idx];
        req_slots_sorted[i] = req_slots[idx];
        req_id_sorted[i] = req_id[idx];
    }
    //device variables after arrays are sorted
    int *d_req_cen_sorted, *d_req_fac_sorted, *d_req_start_sorted, *d_req_slots_sorted;
    hipMalloc(&d_req_cen_sorted, R*sizeof(int));
    hipMalloc(&d_req_fac_sorted, R*sizeof(int));
    hipMalloc(&d_req_start_sorted, R*sizeof(int));
    hipMalloc(&d_req_slots_sorted, R*sizeof(int));
    hipMemcpy(d_req_cen_sorted, req_cen_sorted, R*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_req_fac_sorted, req_fac_sorted, R*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_req_start_sorted, req_start_sorted, R*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_req_slots_sorted, req_slots_sorted, R*sizeof(int), hipMemcpyHostToDevice);

    //*********************************
    // Call the kernels here
    //********************************

    dim3 gridDim(ceil(float(R)/1024),1,1);  
    dim3 blockDim(32,32,1);
    //count the req for same cen and fac
    countkernell<<<gridDim,blockDim>>>(d_req_cen,d_req_fac,d_confreqs,R);
    hipMemcpy(confreqs,d_confreqs,N*max_P*sizeof(int),hipMemcpyDeviceToHost);

    //free d_req_cen and d_req_fac
    hipFree(d_req_cen);
    hipFree(d_req_fac);

    //calculate prefix sum
    prefix[0] = confreqs[0]; // First element of prefix sum array is same as original array
    for (int i = 1; i < N*max_P; i++) {
        prefix[i] = prefix[i - 1] + confreqs[i]; // Calculate prefix sum
    }
    hipMemcpy(d_prefix, prefix, N*max_P * sizeof(int), hipMemcpyHostToDevice);  //store prefix in GPU

    //set the values of d_slotpercenfac in GPU
    dim3 gridDim1(ceil(float(N*max_P*24)/1024),1,1);  
    dim3 blockDim1(32,32,1);
    setarraykernel<<<gridDim1,blockDim1>>>(d_slotspercenfac,d_capacity,N);  //d_capacity stores the capacity of each facility

    //launching successfull request counting kernel with "total possible facility" number of threads
    dim3 gridDim2(ceil(float(N*max_P)/1024),1,1);  
    dim3 blockDim2(32,32,1);
    processRequests<<<gridDim2,blockDim2>>>(R,d_req_cen_sorted,d_req_fac_sorted,d_req_start_sorted,d_req_slots_sorted,d_prefix,d_slotspercenfac,d_succ_reqs,d_tot_reqs,d_succ_cenreqs,N);
    // storeback variables in CPU
    hipMemcpy(succ_reqs, d_succ_reqs,sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(tot_reqs, d_tot_reqs, N* sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(succ_cenreqs, d_succ_cenreqs, N* sizeof(int), hipMemcpyDeviceToHost);

    success=*succ_reqs;  //storing total successfull requests
    fail=R-success;      //storing total failed requests
    char *outputfilename = argv[2]; 
    FILE *outputfilepointer;
    outputfilepointer = fopen(outputfilename,"w");

    fprintf( outputfilepointer, "%d %d\n", success, fail);

    for(int j = 0; j < N; j++)
    {
        fprintf( outputfilepointer, "%d %d\n", succ_cenreqs[j], tot_reqs[j]-succ_cenreqs[j]);
    }
    fclose( inputfilepointer );
    fclose( outputfilepointer );
    hipDeviceSynchronize();
	return 0;
}